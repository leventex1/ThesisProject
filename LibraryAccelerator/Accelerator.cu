﻿#include "MogiAccelerator.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "src/Kernels.h"


namespace_accelerator_start

float* ToDevicePtr(const Tensor2D& tensor, bool copy=true)
{
    float* dPtr;
    size_t size = tensor.GetSize() * sizeof(float);

    hipMalloc(&dPtr, size);
    if (copy)
    {
        hipMemcpy(dPtr, tensor.GetData(), size, hipMemcpyHostToDevice);
    }

    return dPtr;
}

void CopyToHost(Tensor2D& dest, float* deviceSrource)
{
    hipMemcpy(dest.GetData(), deviceSrource, dest.GetSize() * sizeof(float), hipMemcpyDeviceToHost);
}

Tensor2D MatrixMultCUDA(const Tensor2D& left, const Tensor2D& right)
{
    if (left.GetCols() != right.GetRows())
    {
        throw -1;
    }

    Tensor2D res(left.GetRows(), right.GetCols());

    float* dA = ToDevicePtr(left);
    float* dB = ToDevicePtr(right);
    float* dC = ToDevicePtr(res, false);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((right.GetCols() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (left.GetRows()  + threadsPerBlock.y - 1) / threadsPerBlock.y);

    MatrixMultKernel<< <blocksPerGrid, threadsPerBlock >> > (dA, dB, dC, left.GetRows(), left.GetCols(), right.GetCols());

    CopyToHost(res, dC);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return res;
}

Tensor2D MatrixMultRightTransposeCUDA(const Tensor2D& left, const Tensor2D& right)
{
    if (left.GetCols() != right.GetCols())
    {
        throw -1;
    }

    Tensor2D res(left.GetRows(), right.GetRows());

    float* dA = ToDevicePtr(left);
    float* dB = ToDevicePtr(right);
    float* dC = ToDevicePtr(res, false);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((right.GetRows() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (left.GetRows() + threadsPerBlock.y - 1) / threadsPerBlock.y);

    MatrixMultRightTranposeKernel << <blocksPerGrid, threadsPerBlock >> > (dA, dB, dC, left.GetRows(), left.GetCols(), right.GetRows());

    CopyToHost(res, dC);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return res;
}

Tensor2D MatrixMultLeftTransposeCUDA(const Tensor2D& left, const Tensor2D& right)
{
    if (left.GetRows() != right.GetRows())
    {
        throw - 1;
    }

    Tensor2D res(left.GetCols(), right.GetCols());

    float* dA = ToDevicePtr(left);
    float* dB = ToDevicePtr(right);
    float* dC = ToDevicePtr(res, false);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((right.GetCols() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (left.GetCols() + threadsPerBlock.y - 1) / threadsPerBlock.y);

    MatrixMultLeftTranposeKernel<< <blocksPerGrid, threadsPerBlock >> > (dA, dB, dC, left.GetRows(), left.GetCols(), right.GetCols());

    CopyToHost(res, dC);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return res;
}

namespace_accelerator_end