#include "hip/hip_runtime.h"
#include "Kernels.h"

#include ""


__global__ void MatrixMultKernel(const float* A, const float* B, float* C, int ARows, int ACols, int BCols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < ARows && col < BCols) {
        float sum = 0.0;
        for (int i = 0; i < BCols; ++i) {
            sum += A[row * BCols + i] * B[i * BCols + col];
        }
        C[row * BCols + col] = sum;
    }
}

__global__ void MatrixMultRightTranposeKernel(const float* A, const float* B, float* C, int ARows, int ACols, int BRows)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < ARows && col < BRows) {
        float sum = 0.0f;
        for (int e = 0; e < ACols; ++e) {
            sum += A[row * ACols + e] * B[col * ACols + e]; // Accessing B as if it's transposed
        }
        C[row * BRows + col] = sum;
    }
}

__global__ void MatrixMultLeftTranposeKernel(const float* A, const float* B, float* C, int ARows, int ACols, int BCols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < ACols && col < BCols) { // A is transposed, use ACols for row checks
        float sum = 0.0f;
        for (int e = 0; e < ARows; ++e) { // ARows is used here, reflecting the transposed dimension
            sum += A[e * ACols + row] * B[e * BCols + col]; // Access A as transposed
        }
        C[row * BCols + col] = sum;
    }
}